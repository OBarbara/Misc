/*
 ============================================================================
 Name        : NTTFFTCUDA.cu
 Author      : Owen
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA implementation of integer NTT using CG FFT algorithm
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <stdint.h>
#include <math.h>

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
void ProcessandTime(const int size, const int bpe, const int thread_limit);
void initialize(uint32_t* arr, int size, const int bpe);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)
/**
 * CUDA kernel that computes the integer FFT of an array of unsigned integers
 */
// both of the following constants must be a power of two
// the first part takes care of the first stages before the shuffling and the
// element transfer from shared memory to global memory
__global__ void MulSMNTTKernel1(uint32_t *data, const short gridSizeSh, const short th_sec, short curr_k, short k, int mod, const uint32_t* g, bool repos) {
	const int BLOCK_SIZE = blockDim.x;	// the offset given for the second part of shared memory
	if (threadIdx.x < BLOCK_SIZE){	// for maximum performance, the last threadIdx.xshould be a multiple of 32 (minus 1 since it starts from 0)
		uint32_t tempvar; 	  // a temporary variable to manage subtractions and hold temporary data
		short i = threadIdx.x >> 1;
		short j = threadIdx.x & 1;	// even or odd thread
		// used for twiddle factor indexing
		short l = k - curr_k - 1;
		short m = i & 1;	// helps in indexing for each thread in a block
		short elem;
		short b_twid = blockIdx.x << (l - 1);	// twiddle factor indexing for each block
		if(BLOCK_SIZE >= gridDim.x)
			elem = threadIdx.x+ (blockIdx.x << th_sec) + (((threadIdx.x >> th_sec) * (gridDim.x - 1)) << th_sec);
		else
			elem = blockIdx.x + (threadIdx.x << gridSizeSh);
		short p, tmp, x, offx, x1, twid_1, twid_2, twid_index = 0;
		// sign takes care of the operations for even (addition)
		// and odd (subtraction) threads
		short sign = ((-2) * j) + 1;  // can be 1 or -1
		// temporary array to store intermediate data
		extern __shared__ uint32_t final_temp[];
		extern __shared__ uint32_t temp_data[];

		// copy data from global memory to shared memory
		final_temp[threadIdx.x] = data[elem];
		__syncthreads();	// ensure that the shared memory is properly populated


		// stg is the stage (or epoch)
		// k is the number of stages in the computation
		for(short stg = 1; stg <= curr_k; stg++){
			p = BLOCK_SIZE >> stg;	// this variable will help in indexing
			// indexing
			tmp = i + ((i >> (k - curr_k - stg)) * (1 << (k - curr_k - stg)));
			x = tmp + (j * p);
			offx = BLOCK_SIZE + x;	// use this to index second part of shared memory (temp_data[])
			x1 = x + (sign * p);
			if(repos){
				twid_1 = (blockIdx.x + (i << gridSizeSh)) * (curr_k - stg);		// twiddle for stage 1
				twid_2 = ((l - 1) * ((!m * b_twid) + (m * (b_twid + BLOCK_SIZE))) + (curr_k - l) * (blockIdx.x << l)) * (stg - 1);	// twiddle for stage 2
				// twid_index manages the indexing of the twiddle factors
				twid_index =  twid_1 + twid_2;
			}
			// since the value should be unsigned, a subtraction cannot result in a negative number
			// so we add the modulus to the number being subtracted to prevent that from happening
			tempvar = final_temp[x1] + mod;
			// addition and subtraction is taken care of here
			// modulus is done after addition/subtraction
			temp_data[offx] = (tempvar + (sign * final_temp[x])) % mod;
			// shift by twiddle factor and perform modulus
			if(repos){
				temp_data[offx] <<= j * g[twid_index];
				temp_data[offx] %= mod;
			}
			final_temp[x] = temp_data[offx];
			__syncthreads();
		}
		// shuffle data from shared to global memory
		data[elem] = final_temp[threadIdx.x];
	}
}

// second part of the NTT which takes care of the rest of the stages using constant geometry
__global__ void MulSMCGNTTKernel2(uint32_t *data, const short gridSizeSh, short curr_k, short k, int mod, uint32_t* g, bool repos) {
	unsigned short glob_idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int BLOCK_SIZE = blockDim.x;	// the offset given for the second part of shared memory
	if (threadIdx.x < BLOCK_SIZE){	// for maximum performance, the last threadIdx.xshould be a multiple of 32 (minus 1 since it starts from 0)
		uint32_t tempvar;
		short Ndiv2 = BLOCK_SIZE >> 1;
		short j = threadIdx.x & 1;	// even or odd thread
		// x and y are the input indexes
		short x = threadIdx.x >> 1;
		short y = x + Ndiv2;
		short z = threadIdx.x;	// output index
		short j_sign = -(j << 1) + 1;
		short twid_index;
		// temporary array to store intermediate data
		extern __shared__ uint32_t final_temp[];
		extern __shared__ uint32_t temp_data[];

		// copy data from global memory to shared memory
		final_temp[threadIdx.x] = data[glob_idx];
		__syncthreads();	// ensure that the shared memory is properly populate

		// stg is the stage (or epoch)
		// k is the number of stages in the computation
		for(short stg = curr_k; stg <= k; stg = stg + 2){	// stage increments by 2 each time
			tempvar = final_temp[x] + mod;
			// step 1
			// make the subtraction unable to result in a negative number by
			// forcing the first number to be greater than the second if it is not already


			temp_data[z + BLOCK_SIZE] = (tempvar + (j_sign * final_temp[y])) % mod;
			if(repos){
				twid_index = (x >> (stg - 1)) * (1 << (stg - 1)) * repos;
				temp_data[z + BLOCK_SIZE] <<= j * g[twid_index];
				temp_data[z + BLOCK_SIZE] %= mod;
			}
			__syncthreads();	// this makes sure that the final_temp array is read from before it is written to
			if(stg != k){	// all threads will go through one of these branches (no warp divergence here)
				// calculate twiddle factor index
				twid_index = (x >> stg) * (1 << stg);;
				tempvar = temp_data[x + BLOCK_SIZE] + mod;

				// step 2
				final_temp[z] = (tempvar + (j_sign * temp_data[y + BLOCK_SIZE])) % mod;
				if(repos)
					final_temp[z] = (final_temp[z] << (j * g[twid_index])) % mod;
			}
			else
				final_temp[z] = temp_data[z + BLOCK_SIZE];
			__syncthreads();
		}

		// write finished data from shared to global memory
		data[glob_idx] = final_temp[threadIdx.x];
	}
}

// kernel for the inverse NTT FFT using constant geometry
__global__ void MulSMCGINTTKernel1(uint32_t *data, const int NTT_SIZE, const short BITS_PER_ELEM, const short gridSizeSh, short curr_k, short k, int mod, uint32_t* g, bool repos, bool fin) {
	unsigned short glob_idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int BLOCK_SIZE = blockDim.x;	// the offset given for the second part of shared memory
	if (threadIdx.x < BLOCK_SIZE){	// for maximum performance, the last threadIdx.x should be a multiple of 32 (minus 1 since it starts from 0)
		uint32_t tempz1;
		short Ndiv2 = BLOCK_SIZE >> 1;
		short j = threadIdx.x & 1;	// even or odd thread
		// x and y are the input indices
		short x = threadIdx.x >> 1;
		short y = x + Ndiv2;
		short out = (x * !j) + (y * j);	// choose y or j
		short z = threadIdx.x - j;	// input index
		short j_sign = -(j << 1) + 1;
		short twid_index;
		// temporary array to store intermediate data
		extern __shared__ uint32_t final_temp[];
		extern __shared__ uint32_t temp_data[];

		// copy data from global memory to shared memory
		final_temp[threadIdx.x] = data[glob_idx];
		__syncthreads();	// ensure that the shared memory is properly populated

		// stg is the stage (or epoch)
		// k is the number of stages in the computation
		for(short stg = k; stg >= curr_k; stg = stg - 2){	// stage increments by 2 each time
			// make the subtraction unable to result in a negative number by
			// forcing the first number to be greater than the second if it is not already
			tempz1 = final_temp[z] + mod;
			// step 1

			if(repos){
				twid_index = (x >> (stg - 1)) * (1 << (stg - 1));
				temp_data[out + BLOCK_SIZE] = (tempz1 + j_sign * ((final_temp[z + 1] << g[twid_index]) % mod)) % mod;
			}
			else
				temp_data[out + BLOCK_SIZE] = (tempz1 + j_sign * final_temp[z + 1]) % mod;
			__syncthreads();	// this makes sure that the final_temp array is read from before it is written to
			if(stg != curr_k){	// all threads will go through one of these branches (no warp divergence here)
				tempz1 = temp_data[z + BLOCK_SIZE] + mod;

				// step 2
				if(repos){
					// calculate twiddle factor index
					twid_index = (x >> (stg - 2)) * (1 << (stg - 2));
					final_temp[out] = (tempz1 + j_sign * ((temp_data[z + BLOCK_SIZE + 1] << g[twid_index]) % mod)) % mod;
				}
				else
					final_temp[out] = (tempz1 + j_sign * temp_data[z + BLOCK_SIZE + 1]) % mod;
			}
			else
				final_temp[out] = temp_data[out + BLOCK_SIZE];
			__syncthreads();
		}

		// no thread divergence here also
		if(fin){	// if there is only 1 SM finish at this kernel
			// divide each element by N
			short t = (BITS_PER_ELEM << 1) - k; // convert division into multiplication
			uint32_t ls, rs;
			short size = sizeof(uint32_t) << 3; // multiply 4 bytes by 8 in this case (32 bits)
			bool rt_shift = false;	// should the shift be to the right and rotate bits?
			if(t < 0){	// if t is negative, all shifts are to the right
				uint32_t mask = t >> (size - 1);     // make a mask of the sign bit
				t ^= mask;                   // toggle the bits
				t += mask & 1;               // add one
				rt_shift = true;
			}
			short temp_sh;
			short shift = size - (BITS_PER_ELEM + 1);
			// shifting will be done in an optimized manner
			if (shift > t)
				shift = t;	// if t is less than the maximum shift amount, then assign the shift amount to be t
			temp_sh = shift;
			for (short m = 0; m < t; m += shift) {
				shift = temp_sh;	// assign the shift value from the previous iteration

				// there is no thread divergence here since all threads execute the same branch
				if(!rt_shift){	// normal left shift
					final_temp[threadIdx.x] <<= shift;
				}
				else{	// right shift and bit rotation
					rs = final_temp[threadIdx.x] >> shift;
					ls = final_temp[threadIdx.x] << (size - shift);
					final_temp[threadIdx.x] = rs | ls;
				}
				final_temp[threadIdx.x] %= mod;
				if (t - shift <= (BITS_PER_ELEM - 1))
					temp_sh = t - shift;	// all remaining shifts will be done after modulus
			}

			// copy final data from shared memory to global memory
			if(repos)
				data[(NTT_SIZE - glob_idx) % NTT_SIZE] = final_temp[threadIdx.x];
			else
				data[glob_idx] = final_temp[threadIdx.x];
		}
		else{
			// copy from shared memory to global memory in a coalesced manner
			data[glob_idx] = final_temp[threadIdx.x];
		}
	}
}

// shuffles data from global memory to shared memory and computes last set of stages
__global__ void MulSMINTTKernel2(uint32_t *data, const int NTT_SIZE, const short BITS_PER_ELEM, const short gridSizeSh, const short th_sec, short curr_k, short k, int mod, const uint32_t* g, bool repos) {
	const int BLOCK_SIZE = NTT_SIZE >> gridSizeSh;	// the offset given for the second part of shared memory
	if (threadIdx.x < BLOCK_SIZE){	// for maximum performance, the last threadIdx.xshould be a multiple of 32 (minus 1 since it starts from 0)
		uint32_t tempvar; 	  // a temporary variable to manage subtractions and hold temporary data
		short i = threadIdx.x >> 1;
		short j = threadIdx.x & 1;	// even or odd thread
		short l = k - curr_k - 1;
		short m = i & 1;
		short elem;
		short b_twid = blockIdx.x << (l - 1);	// twiddle factor indexing for each block
		if(BLOCK_SIZE >= gridDim.x)
			elem = threadIdx.x+ (blockIdx.x << th_sec) + (((threadIdx.x >> th_sec) * (gridDim.x - 1)) << th_sec);
		else
			elem = blockIdx.x + (threadIdx.x << gridSizeSh);
		short x, tmp, p, offx, x1, twid_1, twid_2, twid_index;
		// sign takes care of the operations for even (addition)
		// and odd (subtraction) threads
		short sign = ((-2) * j) + 1;  // can be 1 or -1
		// temporary array to store intermediate data
		extern __shared__ uint32_t final_temp[];
		extern __shared__ uint32_t temp_data[];

		// copy data from global memory to shared memory
		final_temp[threadIdx.x] = data[elem];
		__syncthreads();	// ensure that the shared memory is properly populated
		// stg is the stage (or epoch)
		// k is the number of stages in the computation
		for(short stg = curr_k; stg >= 1; stg--){
			p = BLOCK_SIZE >> stg;	// this variable will help in indexing
			// indexing
			tmp = i + ((i >> (k - curr_k - stg)) * (1 << (k - curr_k - stg)));
			x = tmp + (j * p);
			offx = BLOCK_SIZE + x;
			x1 = x + (sign * p);
			if(repos){
				twid_1 = (blockIdx.x + (i << gridSizeSh)) * (curr_k - stg);		// twiddle for stage 1
				twid_2 = ((l - 1) * ((!m * b_twid) + (m * (b_twid + BLOCK_SIZE))) + (curr_k - l) * (blockIdx.x << l)) * (stg - 1);	// twiddle for stage 2

				// twid_index manages the indexing of the twiddle factors
				twid_index =  twid_1 + twid_2;
				// shift by twiddle factor and perform modulus
				tempvar = (final_temp[x1] << (!j * g[twid_index])) % mod;
				temp_data[offx] = final_temp[x] << (j * g[twid_index]);
				temp_data[offx] %= mod;
			}
			else{
				tempvar = final_temp[x1];
				temp_data[offx] = final_temp[x];
			}

			// since the value should be unsigned, a subtraction cannot result in a negative number
			// so we add the modulus to the number being subtracted to prevent that from happening
			tempvar += mod;
			// addition and subtraction is taken care of here
			// modulus is done after addition/subtraction
			temp_data[offx] = (tempvar + (sign * temp_data[offx])) % mod;
			final_temp[x] = temp_data[offx];
			// new data is ready for next stage
			__syncthreads();
		}

		// divide each element by N
		short t = (BITS_PER_ELEM << 1) - k; // convert division into multiplication
		uint32_t ls, rs;
		short size = sizeof(uint32_t) << 3; // multiply 4 bytes by 8 in this case (32 bits)
		bool rt_shift = false;	// should the shift be to the right and rotate bits?
		if(t < 0){	// if t is negative, all shifts are to the right
			uint32_t mask = t >> (size - 1);     // make a mask of the sign bit
			t ^= mask;                   // toggle the bits
			t += mask & 1;               // add one
			rt_shift = true;
		}
		short temp_sh;
		short shift = size - (BITS_PER_ELEM + 1);
		// shifting will be done in an optimized manner
		if (shift > t)
			shift = t;	// if t is less than the maximum shift amount, then assign the shift amount to be t
		temp_sh = shift;
		for (m = 0; m < t; m += shift) {
			shift = temp_sh;	// assign the shift value from the previous iteration

			// there is no thread divergence here since all threads execute the same branch
			if(!rt_shift){	// normal left shift
				final_temp[threadIdx.x] <<= shift;
			}
			else{	// right shift and bit rotation
				rs = final_temp[threadIdx.x] >> shift;
				ls = final_temp[threadIdx.x] << (size - shift);
				final_temp[threadIdx.x] = rs | ls;
			}
			final_temp[threadIdx.x] %= mod;
			if (t - shift <= (BITS_PER_ELEM - 1))
				temp_sh = t - shift;	// all remaining shifts will be done after modulus
		}

		// copy final data from shared memory to global memory
		if(repos)
			data[(NTT_SIZE - elem) % NTT_SIZE] = final_temp[threadIdx.x];
		else
			data[elem] = final_temp[threadIdx.x];
	}
}

/*! \name GPU function caller and process timing function */
void ProcessandTime(const int size, const int bpe, const int thlimit){
	// define NTT variables
	double intervalNTT = 0, intervalINTT = 0;
	float NTTms, iNTTms;
	hipEvent_t NTTstart, NTTstop, INTTstart, INTTstop;
	hipEventCreate(&NTTstart);
	hipEventCreate(&NTTstop);
	hipEventCreate(&INTTstart);
	hipEventCreate(&INTTstop);

	short kt = log2((double)size);
	int i;	// index for traversing through the arrays
	int modulus = (1 << bpe) + 1; // prime number for modulo arithmetic
	int rt_unity = 2 * bpe / size; // calculate # bits for root of unity
	// these flags determine whether the IFFT repositions the elements
	// and whether the result of the IFFT matches the input array or not, respectively
	bool repos_flag = true, flag = true;
	int runs = 1E3;	// number of times each kernel function is executed
	uint32_t *in = new uint32_t[size];
	uint32_t *NTT_out = new uint32_t[size];
	uint32_t *INTT_out = new uint32_t[size];
	uint32_t* gpuNTTData, *gpuINTTData, *gpuTwid;
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuNTTData, sizeof(uint32_t) * size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuTwid, sizeof(uint32_t)*size / 2));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuINTTData, sizeof(uint32_t)*size));
	uint32_t twiddle[size / 2];	// pre-compute twiddle factor array
	if(!rt_unity)
		repos_flag = false;

	for(int i = 0; i < size / 2; i++){
		twiddle[i] = rt_unity * i;
	}
	int blk_size;
	bool isSingleSM = false;
	if(size <= thlimit){
		blk_size = size;
		isSingleSM = true;
	}
	else
		blk_size = thlimit;
	static const int BLOCK_SIZE = blk_size;	// amount of threads in each block
	const int blockCount = (size) / BLOCK_SIZE;	// amount of blocks in a grid
	short k = log2((double)blockCount);	// first kernel covers first k stages only out of kt for FFT, and vice-versa for IFFT kernels
	const short gridSizeSh = logf(blockCount) / logf(2);
	short coal_segment = BLOCK_SIZE >> gridSizeSh;
	coal_segment = log2((float)coal_segment);
	std::cout << "Launching kernels with " << blockCount << " block(s), each with " << BLOCK_SIZE << " threads." << std::endl;
	srand(time(NULL));	// generate the seed for the pseudo-random number generator
	for (int j = 0; j < runs; j++) {
		initialize(in, modulus, size);

		CUDA_CHECK_RETURN(hipMemcpy(gpuNTTData, in, sizeof(uint32_t)*size, hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(gpuTwid, twiddle, sizeof(uint32_t)*size/2, hipMemcpyHostToDevice));

		hipEventRecord(NTTstart);
		if(!isSingleSM){
			// computes first k stages and shuffles data from shared memory to global memory
			MulSMNTTKernel1<<<blockCount, BLOCK_SIZE, 2 * BLOCK_SIZE * sizeof(uint32_t)>>> (gpuNTTData, gridSizeSh, coal_segment, k, kt, modulus, gpuTwid, repos_flag);
		}
		// shuffles data from global memory to shared memory and computes last set of stages
		MulSMCGNTTKernel2<<<blockCount, BLOCK_SIZE, 2 * BLOCK_SIZE * sizeof(uint32_t)>>> (gpuNTTData, gridSizeSh, k + 1, kt, modulus, gpuTwid, repos_flag);
		hipEventRecord(NTTstop);

		CUDA_CHECK_RETURN(hipMemcpy(NTT_out, gpuNTTData, sizeof(uint32_t) * size, hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(gpuINTTData, NTT_out, sizeof(uint32_t)*size, hipMemcpyHostToDevice));
		hipEventSynchronize(NTTstop);
		hipEventElapsedTime(&NTTms, NTTstart, NTTstop);
		hipEventRecord(INTTstart);
		// computes first stages and shuffles data from shared memory to global memory
		MulSMCGINTTKernel1<<<blockCount, BLOCK_SIZE, 2 * BLOCK_SIZE * sizeof(uint32_t)>>> (gpuINTTData, size, bpe, gridSizeSh, k + 1, kt, modulus, gpuTwid, repos_flag, isSingleSM);
		if(!isSingleSM){
			// shuffles data from global memory to shared memory and computes last set of stages
			MulSMINTTKernel2<<<blockCount, BLOCK_SIZE, 2 * BLOCK_SIZE * sizeof(uint32_t)>>> (gpuINTTData, size, bpe, gridSizeSh, coal_segment, k, kt, modulus, gpuTwid, repos_flag);
		}
		hipEventRecord(INTTstop);

		CUDA_CHECK_RETURN(hipMemcpy(INTT_out, gpuINTTData, sizeof(uint32_t)*size, hipMemcpyDeviceToHost));

		hipEventSynchronize(INTTstop);
		hipEventElapsedTime(&iNTTms, INTTstart, INTTstop);

		// total time taken is summed up
		intervalNTT += NTTms;
		intervalINTT += iNTTms;


		for (i = 0; i < size; i++) {
			if (in[i] != INTT_out[i]) {
				flag = false;
				break;
			}
		}

		if (!flag) {
			std::cout << "j = " << j << ", i = " << i << "\nin = " << in[i] << ", out = " << INTT_out[i] << std::endl;
			break;
		}
	}

	if (flag){
		std::cout << "NTT matched. Average time taken for NTT is: " << intervalNTT << " microseconds" << std::endl;
		std::cout << "Average time taken for INTT is: " << intervalINTT << " microseconds" << std::endl;
	}

	// clean up
	hipEventDestroy(NTTstart);
	hipEventDestroy(NTTstop);
	hipEventDestroy(INTTstart);
	hipEventDestroy(INTTstop);
	CUDA_CHECK_RETURN(hipFree(gpuNTTData));
	CUDA_CHECK_RETURN(hipFree(gpuINTTData));
	CUDA_CHECK_RETURN(hipFree(gpuTwid));
	delete[] in;
	delete[] NTT_out;
	delete[] INTT_out;
	return;
}

// @}


/* \name Array initializer function */
void initialize(uint32_t* in, int mod, const int size)
{
	for (int i = 0; i < size; i++)
	{
		in[i] = (uint32_t)(rand() % mod);
	}
}

// @}

int main(int argc, char *argv[])
{
	std::cerr << "NTT FFT with constant geometry" << std::endl;
	const int NTT_SIZE = atoi(argv[1]);
	const int BITS_PER_ELEM = atoi(argv[2]);
	const int TH_LIM = atoi(argv[3]);
	// check if both of the required arguments are powers of two
	double chk1 = log2((double)NTT_SIZE);
	double chk2 = log2((double)BITS_PER_ELEM);
	double chk3 = (double)logf(TH_LIM) / logf(2);
	if (argc != 4 || chk1 != round(chk1) || chk2 != round(chk2) || chk3 != round(chk3) ||  TH_LIM < 0 || TH_LIM > 1024 || BITS_PER_ELEM > 16
				|| NTT_SIZE/TH_LIM > TH_LIM || NTT_SIZE <= 1 || BITS_PER_ELEM <= 1)
	{
	  std::cerr << "Usage: " << argv[0]
			<< " <NTT size> <Bits per element> <thread limit per block> " << std::endl;
	  exit(1);
	}
	std::cout << "Computing..." << std::endl;
	ProcessandTime(NTT_SIZE, BITS_PER_ELEM, TH_LIM);
	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}
